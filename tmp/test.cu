#include <hip/hip_runtime.h>
#include <iostream>

#define uint unsigned int
#define uchar unsigned char
#define ushort unsigned short
#define int64_t long long
#define uint64_t unsigned long long

extern "C" __global__ void conv3(float* __restrict__ data,
				 float* __restrict__ kernel,
				 float* __restrict__ compute) {
  float compute_local[32];
  __shared__ float pad_temp_shared[720];
  __shared__ float kernel_shared[1152];
  float pad_temp_shared_local[10];
  float kernel_shared_local[12];
  for (int yy_c_init = 0; yy_c_init < 8; ++yy_c_init) {
    compute_local[(yy_c_init)] = 0.000000e+00f;
    compute_local[((yy_c_init + 8))] = 0.000000e+00f;
    compute_local[((yy_c_init + 16))] = 0.000000e+00f;
    compute_local[((yy_c_init + 24))] = 0.000000e+00f;
  }
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 6; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((int)threadIdx.z) * 5) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) < 40) {
        if ((((((int)threadIdx.z) * 90) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 720) {
          if (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 90) {
            pad_temp_shared[((((((int)threadIdx.z) * 90) + (((int)threadIdx.x) * 6)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((1 <= ((((int)blockIdx.y) * 8) + (((((int)threadIdx.z) * 5) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) % 10))) && (((((int)blockIdx.y) * 8) + (((((int)threadIdx.z) * 5) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) % 10)) < 257)) && (1 <= ((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)))) && (((((int)blockIdx.x) * 16) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)) < 257)) ? data[((((((((rc_outer * 262144) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) / 10) * 65536)) + (((int)blockIdx.y) * 2048)) + ((((((int)threadIdx.z) * 5) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 18)) % 10) * 256)) + (((int)blockIdx.x) * 16)) + (((((int)threadIdx.x) * 6) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) % 18)) - 257))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 9; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      kernel_shared[((((((int)threadIdx.z) * 144) + (((int)threadIdx.x) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[((((((((int)threadIdx.z) * 1152) + ((((int)threadIdx.x) >> 2) * 288)) + (rc_outer * 36)) + ((((int)threadIdx.x) & 3) * 9)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))];
    }
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
      for (int rx_inner_outer = 0; rx_inner_outer < 3; ++rx_inner_outer) {
        for (int ax2 = 0; ax2 < 10; ++ax2) {
          pad_temp_shared_local[(ax2)] = pad_temp_shared[(((((rc_inner_outer * 180) + (ax2 * 18)) + ((int)threadIdx.x)) + rx_inner_outer))];
        }
        for (int ax21 = 0; ax21 < 3; ++ax21) {
          kernel_shared_local[(ax21)] = kernel_shared[(((((((int)threadIdx.z) * 36) + (rc_inner_outer * 9)) + (ax21 * 3)) + rx_inner_outer))];
          kernel_shared_local[((ax21 + 3))] = kernel_shared[((((((((int)threadIdx.z) * 36) + (rc_inner_outer * 9)) + (ax21 * 3)) + rx_inner_outer) + 288))];
          kernel_shared_local[((ax21 + 6))] = kernel_shared[((((((((int)threadIdx.z) * 36) + (rc_inner_outer * 9)) + (ax21 * 3)) + rx_inner_outer) + 576))];
          kernel_shared_local[((ax21 + 9))] = kernel_shared[((((((((int)threadIdx.z) * 36) + (rc_inner_outer * 9)) + (ax21 * 3)) + rx_inner_outer) + 864))];
        }
        for (int ry_inner_inner = 0; ry_inner_inner < 3; ++ry_inner_inner) {
          for (int yy_c = 0; yy_c < 8; ++yy_c) {
            compute_local[(yy_c)] = (compute_local[(yy_c)] + (pad_temp_shared_local[((yy_c + ry_inner_inner))] * kernel_shared_local[(ry_inner_inner)]));
            compute_local[((yy_c + 8))] = (compute_local[((yy_c + 8))] + (pad_temp_shared_local[((yy_c + ry_inner_inner))] * kernel_shared_local[((ry_inner_inner + 3))]));
            compute_local[((yy_c + 16))] = (compute_local[((yy_c + 16))] + (pad_temp_shared_local[((yy_c + ry_inner_inner))] * kernel_shared_local[((ry_inner_inner + 6))]));
            compute_local[((yy_c + 24))] = (compute_local[((yy_c + 24))] + (pad_temp_shared_local[((yy_c + ry_inner_inner))] * kernel_shared_local[((ry_inner_inner + 9))]));
          }
        }
      }
    }
  }
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 8; ++yy_inner_inner_inner) {
    compute[((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 2048)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)))] = compute_local[(yy_inner_inner_inner)];
    compute[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 2048)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 524288))] = compute_local[((yy_inner_inner_inner + 8))];
    compute[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 2048)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 1048576))] = compute_local[((yy_inner_inner_inner + 16))];
    compute[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 2048)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 16)) + ((int)threadIdx.x)) + 1572864))] = compute_local[((yy_inner_inner_inner + 24))];
  }
}

extern "C" __global__ void conv5(float* __restrict__ data,
				 float* __restrict__ kernel,
				 float* __restrict__ compute) {
  float compute_local[64];
  __shared__ float pad_temp_shared[544];
  __shared__ float kernel_shared[800];
  float pad_temp_shared_local[16];
  float kernel_shared_local[4];
  for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
    for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
      compute_local[(((yy_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 16))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 32))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 48))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 8))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 24))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 40))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 56))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 4))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 20))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 36))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 52))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 12))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 28))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 44))] = 0.000000e+00f;
      compute_local[((((yy_c_init * 2) + xx_c_init) + 60))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner < 5; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) {
      if (((((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) / 68) + ((int)threadIdx.z)) < 8) {
        if ((((((int)threadIdx.z) * 68) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 544) {
          if (((((int)threadIdx.x) * 5) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 68) {
            pad_temp_shared[((((((int)threadIdx.z) * 68) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))] = (((((2 <= ((((int)blockIdx.y) * 4) + ((int)threadIdx.z))) && (((((int)blockIdx.y) * 4) + ((int)threadIdx.z)) < 258)) && (2 <= (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner))) && ((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) < 258)) ? data[((((((((rc_outer * 65536) + (((int)blockIdx.y) * 1024)) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 5)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner) - 514))] : 0.000000e+00f);
          }
        }
      }
    }
    for (int ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 = 0; ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1 < 7; ++ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 25)) < 32) {
        if (((((int)threadIdx.z) * 20) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 5)) < 160) {
          if ((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 800) {
            if (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) < 100) {
              kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 7)) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1))] = kernel[(((((((int)threadIdx.z) * 3200) + ((((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) / 25) * 800)) + (rc_outer * 25)) + (((((int)threadIdx.x) * 7) + ax0_ax1_fused_ax2_fused_ax3_fused_inner_inner_inner1) % 25)))];
            }
          }
        }
      }
    }
    __syncthreads();
    for (int ry_inner_outer = 0; ry_inner_outer < 5; ++ry_inner_outer) {
      for (int rx_inner_outer = 0; rx_inner_outer < 5; ++rx_inner_outer) {
        for (int ax2 = 0; ax2 < 2; ++ax2) {
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(((ax2 * 2) + ax3))] = pad_temp_shared[((((((ax2 * 68) + (ry_inner_outer * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 8))] = pad_temp_shared[(((((((ax2 * 68) + (ry_inner_outer * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 136))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 4))] = pad_temp_shared[(((((((ax2 * 68) + (ry_inner_outer * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 32))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 12))] = pad_temp_shared[(((((((ax2 * 68) + (ry_inner_outer * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_inner_outer) + 168))];
          }
        }
        kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 25) + (ry_inner_outer * 5)) + rx_inner_outer))];
        kernel_shared_local[(1)] = kernel_shared[(((((((int)threadIdx.z) * 25) + (ry_inner_outer * 5)) + rx_inner_outer) + 200))];
        kernel_shared_local[(2)] = kernel_shared[(((((((int)threadIdx.z) * 25) + (ry_inner_outer * 5)) + rx_inner_outer) + 400))];
        kernel_shared_local[(3)] = kernel_shared[(((((((int)threadIdx.z) * 25) + (ry_inner_outer * 5)) + rx_inner_outer) + 600))];
        for (int yy_c = 0; yy_c < 2; ++yy_c) {
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute_local[(((yy_c * 2) + xx_c))] = (compute_local[(((yy_c * 2) + xx_c))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(0)]));
            compute_local[((((yy_c * 2) + xx_c) + 16))] = (compute_local[((((yy_c * 2) + xx_c) + 16))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(1)]));
            compute_local[((((yy_c * 2) + xx_c) + 32))] = (compute_local[((((yy_c * 2) + xx_c) + 32))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(2)]));
            compute_local[((((yy_c * 2) + xx_c) + 48))] = (compute_local[((((yy_c * 2) + xx_c) + 48))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(3)]));
            compute_local[((((yy_c * 2) + xx_c) + 8))] = (compute_local[((((yy_c * 2) + xx_c) + 8))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(0)]));
            compute_local[((((yy_c * 2) + xx_c) + 24))] = (compute_local[((((yy_c * 2) + xx_c) + 24))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(1)]));
            compute_local[((((yy_c * 2) + xx_c) + 40))] = (compute_local[((((yy_c * 2) + xx_c) + 40))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(2)]));
            compute_local[((((yy_c * 2) + xx_c) + 56))] = (compute_local[((((yy_c * 2) + xx_c) + 56))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(3)]));
            compute_local[((((yy_c * 2) + xx_c) + 4))] = (compute_local[((((yy_c * 2) + xx_c) + 4))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(0)]));
            compute_local[((((yy_c * 2) + xx_c) + 20))] = (compute_local[((((yy_c * 2) + xx_c) + 20))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(1)]));
            compute_local[((((yy_c * 2) + xx_c) + 36))] = (compute_local[((((yy_c * 2) + xx_c) + 36))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(2)]));
            compute_local[((((yy_c * 2) + xx_c) + 52))] = (compute_local[((((yy_c * 2) + xx_c) + 52))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(3)]));
            compute_local[((((yy_c * 2) + xx_c) + 12))] = (compute_local[((((yy_c * 2) + xx_c) + 12))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(0)]));
            compute_local[((((yy_c * 2) + xx_c) + 28))] = (compute_local[((((yy_c * 2) + xx_c) + 28))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(1)]));
            compute_local[((((yy_c * 2) + xx_c) + 44))] = (compute_local[((((yy_c * 2) + xx_c) + 44))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(2)]));
            compute_local[((((yy_c * 2) + xx_c) + 60))] = (compute_local[((((yy_c * 2) + xx_c) + 60))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(3)]));
          }
        }
      }
    }
  }
  for (int yy_inner_inner_inner = 0; yy_inner_inner_inner < 2; ++yy_inner_inner_inner) {
    for (int xx_inner_inner_inner = 0; xx_inner_inner_inner < 2; ++xx_inner_inner_inner) {
      compute[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner))] = compute_local[(((yy_inner_inner_inner * 2) + xx_inner_inner_inner))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 524288))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 16))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1048576))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 32))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1572864))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 48))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 512))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 8))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 524800))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 24))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1049088))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 40))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1573376))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 56))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 32))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 4))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 524320))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 20))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1048608))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 36))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1572896))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 52))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 544))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 12))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 524832))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 28))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1049120))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 44))];
      compute[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_inner_inner_inner * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_inner_inner_inner) + 1573408))] = compute_local[((((yy_inner_inner_inner * 2) + xx_inner_inner_inner) + 60))];
    }
  }
}

extern "C" __global__ void conv_partially_fused(float* __restrict__ data,
						float* __restrict__ kernel5,
						float* __restrict__ kernel3,
						float* __restrict__ compute5,
						float* __restrict__ compute3) {
  float compute5_local[64];
  float compute3_local[64];
  __shared__ float pad_temp_shared[544];
  __shared__ float kernel5_shared[800];
  __shared__ float kernel3_shared[288];
  float pad_temp_shared_local[16];
  float kernel_shared_local[4];
  for (int yy_c_init = 0; yy_c_init < 2; ++yy_c_init) {
    for (int xx_c_init = 0; xx_c_init < 2; ++xx_c_init) {
      compute5_local[(((yy_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 16))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 32))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 48))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 8))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 24))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 40))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 56))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 4))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 20))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 36))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 52))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 12))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 28))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 44))] = 0.000000e+00f;
      compute5_local[((((yy_c_init * 2) + xx_c_init) + 60))] = 0.000000e+00f;

      compute3_local[(((yy_c_init * 2) + xx_c_init))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 16))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 32))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 48))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 8))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 24))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 40))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 56))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 4))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 20))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 36))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 52))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 12))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 28))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 44))] = 0.000000e+00f;
      compute3_local[((((yy_c_init * 2) + xx_c_init) + 60))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 32; ++rc_outer) {
    __syncthreads();
    // Load image to shared
    for (int ax_ffiii = 0; ax_ffiii < 5; ++ax_ffiii) {
      if (((((((int)threadIdx.x) * 5) + ax_ffiii) / 68) + ((int)threadIdx.z)) < 8) {
        if ((((((int)threadIdx.z) * 68) + (((int)threadIdx.x) * 5)) + ax_ffiii) < 544) {
          if (((((int)threadIdx.x) * 5) + ax_ffiii) < 68) {
            pad_temp_shared[((((((int)threadIdx.z) * 68) + (((int)threadIdx.x) * 5)) + ax_ffiii))] = (((((2 <= ((((int)blockIdx.y) * 4) + ((int)threadIdx.z))) && (((((int)blockIdx.y) * 4) + ((int)threadIdx.z)) < 258)) && (2 <= (((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 5)) + ax_ffiii))) && ((((((int)blockIdx.x) * 64) + (((int)threadIdx.x) * 5)) + ax_ffiii) < 258)) ? data[((((((((rc_outer * 65536) + (((int)blockIdx.y) * 1024)) + (((int)threadIdx.z) * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 5)) + ax_ffiii) - 514))] : 0.000000e+00f);
          }
        }
      }
    }

    // Load 5x5 kernel to shared
    for (int ax_ffiii1 = 0; ax_ffiii1 < 7; ++ax_ffiii1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 7) + ax_ffiii1) / 25)) < 32) {
        if (((((int)threadIdx.z) * 20) + (((((int)threadIdx.x) * 7) + ax_ffiii1) / 5)) < 160) {
          if ((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 7)) + ax_ffiii1) < 800) {
            if (((((int)threadIdx.x) * 7) + ax_ffiii1) < 100) {
              kernel5_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 7)) + ax_ffiii1))] = kernel5[(((((((int)threadIdx.z) * 3200) + ((((((int)threadIdx.x) * 7) + ax_ffiii1) / 25) * 800)) + (rc_outer * 25)) + (((((int)threadIdx.x) * 7) + ax_ffiii1) % 25)))];
            }
          }
        }
      }
    }

    // Load 3x3 kernel to shared
    for (int ax_ffiii1 = 0; ax_ffiii1 < 3; ++ax_ffiii1) {
      if (((((int)threadIdx.z) * 4) + (((((int)threadIdx.x) * 3) + ax_ffiii1) / 9)) < 32) {
        if (((((int)threadIdx.z) * 12) + (((((int)threadIdx.x) * 3) + ax_ffiii1) / 3)) < 96) {
          if ((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 3)) + ax_ffiii1) < 288) {
            if (((((int)threadIdx.x) * 3) + ax_ffiii1) < 36) {
              kernel3_shared[((((((int)threadIdx.z) * 36) + (((int)threadIdx.x) * 3)) + ax_ffiii1))] = kernel3[(((((((int)threadIdx.z) * 1152) + ((((((int)threadIdx.x) * 3) + ax_ffiii1) / 25) * 288)) + (rc_outer * 9)) + (((((int)threadIdx.x) * 3) + ax_ffiii1) % 9)))];
            }
          }
        }
      }
    }
    __syncthreads();

    // 5x5 reduction in registers
    for (int ry_io = 0; ry_io < 5; ++ry_io) {
      for (int rx_io = 0; rx_io < 5; ++rx_io) {
        for (int ax2 = 0; ax2 < 2; ++ax2) {
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(((ax2 * 2) + ax3))] = pad_temp_shared[((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 8))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 136))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 4))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 32))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 12))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 168))];
          }
        }
        kernel_shared_local[(0)] = kernel5_shared[((((((int)threadIdx.z) * 25) + (ry_io * 5)) + rx_io))];
        kernel_shared_local[(1)] = kernel5_shared[(((((((int)threadIdx.z) * 25) + (ry_io * 5)) + rx_io) + 200))];
        kernel_shared_local[(2)] = kernel5_shared[(((((((int)threadIdx.z) * 25) + (ry_io * 5)) + rx_io) + 400))];
        kernel_shared_local[(3)] = kernel5_shared[(((((((int)threadIdx.z) * 25) + (ry_io * 5)) + rx_io) + 600))];
        for (int yy_c = 0; yy_c < 2; ++yy_c) {
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute5_local[(((yy_c * 2) + xx_c))] = (compute5_local[(((yy_c * 2) + xx_c))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(0)]));
            compute5_local[((((yy_c * 2) + xx_c) + 16))] = (compute5_local[((((yy_c * 2) + xx_c) + 16))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(1)]));
            compute5_local[((((yy_c * 2) + xx_c) + 32))] = (compute5_local[((((yy_c * 2) + xx_c) + 32))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(2)]));
            compute5_local[((((yy_c * 2) + xx_c) + 48))] = (compute5_local[((((yy_c * 2) + xx_c) + 48))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(3)]));
            compute5_local[((((yy_c * 2) + xx_c) + 8))] = (compute5_local[((((yy_c * 2) + xx_c) + 8))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(0)]));
            compute5_local[((((yy_c * 2) + xx_c) + 24))] = (compute5_local[((((yy_c * 2) + xx_c) + 24))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(1)]));
            compute5_local[((((yy_c * 2) + xx_c) + 40))] = (compute5_local[((((yy_c * 2) + xx_c) + 40))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(2)]));
            compute5_local[((((yy_c * 2) + xx_c) + 56))] = (compute5_local[((((yy_c * 2) + xx_c) + 56))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(3)]));
            compute5_local[((((yy_c * 2) + xx_c) + 4))] = (compute5_local[((((yy_c * 2) + xx_c) + 4))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(0)]));
            compute5_local[((((yy_c * 2) + xx_c) + 20))] = (compute5_local[((((yy_c * 2) + xx_c) + 20))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(1)]));
            compute5_local[((((yy_c * 2) + xx_c) + 36))] = (compute5_local[((((yy_c * 2) + xx_c) + 36))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(2)]));
            compute5_local[((((yy_c * 2) + xx_c) + 52))] = (compute5_local[((((yy_c * 2) + xx_c) + 52))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(3)]));
            compute5_local[((((yy_c * 2) + xx_c) + 12))] = (compute5_local[((((yy_c * 2) + xx_c) + 12))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(0)]));
            compute5_local[((((yy_c * 2) + xx_c) + 28))] = (compute5_local[((((yy_c * 2) + xx_c) + 28))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(1)]));
            compute5_local[((((yy_c * 2) + xx_c) + 44))] = (compute5_local[((((yy_c * 2) + xx_c) + 44))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(2)]));
            compute5_local[((((yy_c * 2) + xx_c) + 60))] = (compute5_local[((((yy_c * 2) + xx_c) + 60))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(3)]));
          }
        }
      }
    }

    // 3x3 reduction in registers
    for (int ry_io = 0; ry_io < 3; ++ry_io) {
      for (int rx_io = 0; rx_io < 3; ++rx_io) {
        for (int ax2 = 0; ax2 < 2; ++ax2) {
          for (int ax3 = 0; ax3 < 2; ++ax3) {
            pad_temp_shared_local[(((ax2 * 2) + ax3))] = pad_temp_shared[((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 8))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 136))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 4))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 32))];
            pad_temp_shared_local[((((ax2 * 2) + ax3) + 12))] = pad_temp_shared[(((((((ax2 * 68) + (ry_io * 68)) + (((int)threadIdx.x) * 2)) + ax3) + rx_io) + 168))];
          }
        }
        kernel_shared_local[(0)] = kernel3_shared[((((((int)threadIdx.z) * 9) + (ry_io * 3)) + rx_io))];
        kernel_shared_local[(1)] = kernel3_shared[(((((((int)threadIdx.z) * 9) + (ry_io * 3)) + rx_io) + 72))];
        kernel_shared_local[(2)] = kernel3_shared[(((((((int)threadIdx.z) * 9) + (ry_io * 3)) + rx_io) + 144))];
        kernel_shared_local[(3)] = kernel3_shared[(((((((int)threadIdx.z) * 9) + (ry_io * 3)) + rx_io) + 216))];
        for (int yy_c = 0; yy_c < 2; ++yy_c) {
          for (int xx_c = 0; xx_c < 2; ++xx_c) {
            compute3_local[(((yy_c * 2) + xx_c))] = (compute3_local[(((yy_c * 2) + xx_c))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(0)]));
            compute3_local[((((yy_c * 2) + xx_c) + 16))] = (compute3_local[((((yy_c * 2) + xx_c) + 16))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(1)]));
            compute3_local[((((yy_c * 2) + xx_c) + 32))] = (compute3_local[((((yy_c * 2) + xx_c) + 32))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(2)]));
            compute3_local[((((yy_c * 2) + xx_c) + 48))] = (compute3_local[((((yy_c * 2) + xx_c) + 48))] + (pad_temp_shared_local[(((yy_c * 2) + xx_c))] * kernel_shared_local[(3)]));
            compute3_local[((((yy_c * 2) + xx_c) + 8))] = (compute3_local[((((yy_c * 2) + xx_c) + 8))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(0)]));
            compute3_local[((((yy_c * 2) + xx_c) + 24))] = (compute3_local[((((yy_c * 2) + xx_c) + 24))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(1)]));
            compute3_local[((((yy_c * 2) + xx_c) + 40))] = (compute3_local[((((yy_c * 2) + xx_c) + 40))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(2)]));
            compute3_local[((((yy_c * 2) + xx_c) + 56))] = (compute3_local[((((yy_c * 2) + xx_c) + 56))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 8))] * kernel_shared_local[(3)]));
            compute3_local[((((yy_c * 2) + xx_c) + 4))] = (compute3_local[((((yy_c * 2) + xx_c) + 4))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(0)]));
            compute3_local[((((yy_c * 2) + xx_c) + 20))] = (compute3_local[((((yy_c * 2) + xx_c) + 20))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(1)]));
            compute3_local[((((yy_c * 2) + xx_c) + 36))] = (compute3_local[((((yy_c * 2) + xx_c) + 36))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(2)]));
            compute3_local[((((yy_c * 2) + xx_c) + 52))] = (compute3_local[((((yy_c * 2) + xx_c) + 52))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 4))] * kernel_shared_local[(3)]));
            compute3_local[((((yy_c * 2) + xx_c) + 12))] = (compute3_local[((((yy_c * 2) + xx_c) + 12))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(0)]));
            compute3_local[((((yy_c * 2) + xx_c) + 28))] = (compute3_local[((((yy_c * 2) + xx_c) + 28))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(1)]));
            compute3_local[((((yy_c * 2) + xx_c) + 44))] = (compute3_local[((((yy_c * 2) + xx_c) + 44))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(2)]));
            compute3_local[((((yy_c * 2) + xx_c) + 60))] = (compute3_local[((((yy_c * 2) + xx_c) + 60))] + (pad_temp_shared_local[((((yy_c * 2) + xx_c) + 12))] * kernel_shared_local[(3)]));
          }
        }
      }
    }
  }

  // Store to global
  for (int yy_iii = 0; yy_iii < 2; ++yy_iii) {
    for (int xx_iii = 0; xx_iii < 2; ++xx_iii) {
      compute5[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii))] = compute5_local[(((yy_iii * 2) + xx_iii))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524288))] = compute5_local[((((yy_iii * 2) + xx_iii) + 16))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1048576))] = compute5_local[((((yy_iii * 2) + xx_iii) + 32))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1572864))] = compute5_local[((((yy_iii * 2) + xx_iii) + 48))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 512))] = compute5_local[((((yy_iii * 2) + xx_iii) + 8))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524800))] = compute5_local[((((yy_iii * 2) + xx_iii) + 24))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1049088))] = compute5_local[((((yy_iii * 2) + xx_iii) + 40))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1573376))] = compute5_local[((((yy_iii * 2) + xx_iii) + 56))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 32))] = compute5_local[((((yy_iii * 2) + xx_iii) + 4))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524320))] = compute5_local[((((yy_iii * 2) + xx_iii) + 20))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1048608))] = compute5_local[((((yy_iii * 2) + xx_iii) + 36))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1572896))] = compute5_local[((((yy_iii * 2) + xx_iii) + 52))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 544))] = compute5_local[((((yy_iii * 2) + xx_iii) + 12))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524832))] = compute5_local[((((yy_iii * 2) + xx_iii) + 28))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1049120))] = compute5_local[((((yy_iii * 2) + xx_iii) + 44))];
      compute5[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1573408))] = compute5_local[((((yy_iii * 2) + xx_iii) + 60))];


      compute3[(((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii))] = compute3_local[(((yy_iii * 2) + xx_iii))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524288))] = compute3_local[((((yy_iii * 2) + xx_iii) + 16))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1048576))] = compute3_local[((((yy_iii * 2) + xx_iii) + 32))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1572864))] = compute3_local[((((yy_iii * 2) + xx_iii) + 48))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 512))] = compute3_local[((((yy_iii * 2) + xx_iii) + 8))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524800))] = compute3_local[((((yy_iii * 2) + xx_iii) + 24))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1049088))] = compute3_local[((((yy_iii * 2) + xx_iii) + 40))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1573376))] = compute3_local[((((yy_iii * 2) + xx_iii) + 56))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 32))] = compute3_local[((((yy_iii * 2) + xx_iii) + 4))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524320))] = compute3_local[((((yy_iii * 2) + xx_iii) + 20))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1048608))] = compute3_local[((((yy_iii * 2) + xx_iii) + 36))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1572896))] = compute3_local[((((yy_iii * 2) + xx_iii) + 52))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 544))] = compute3_local[((((yy_iii * 2) + xx_iii) + 12))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 524832))] = compute3_local[((((yy_iii * 2) + xx_iii) + 28))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1049120))] = compute3_local[((((yy_iii * 2) + xx_iii) + 44))];
      compute3[((((((((((int)threadIdx.z) * 65536) + (((int)blockIdx.y) * 1024)) + (yy_iii * 256)) + (((int)blockIdx.x) * 64)) + (((int)threadIdx.x) * 2)) + xx_iii) + 1573408))] = compute3_local[((((yy_iii * 2) + xx_iii) + 60))];
    }
  }
}

extern "C" __global__ void conv_partially_fused_old(float* __restrict__ data,
						    float* __restrict__ kernel5,
						    float* __restrict__ kernel3,
						    float* __restrict__ compute5,
						    float* __restrict__ compute3) {
  float compute5_local[64];
  float compute3_local[64];
  __shared__ float pad_temp_shared[1600];
  __shared__ float kernel5_shared[320];
  __shared__ float kernel3_shared[192];
  float pad_temp_shared_local[16];
  float kernel_shared_local[4];
  for (int ff_c_init = 0; ff_c_init < 4; ++ff_c_init) {
    for (int xx_c_init = 0; xx_c_init < 4; ++xx_c_init) {
      compute5_local[(((ff_c_init * 4) + xx_c_init))] = 0.000000e+00f;
      compute5_local[((((ff_c_init * 4) + xx_c_init) + 16))] = 0.000000e+00f;
      compute5_local[((((ff_c_init * 4) + xx_c_init) + 32))] = 0.000000e+00f;
      compute5_local[((((ff_c_init * 4) + xx_c_init) + 48))] = 0.000000e+00f;
      compute3_local[(((ff_c_init * 4) + xx_c_init))] = 0.000000e+00f;
      compute3_local[((((ff_c_init * 4) + xx_c_init) + 16))] = 0.000000e+00f;
      compute3_local[((((ff_c_init * 4) + xx_c_init) + 32))] = 0.000000e+00f;
      compute3_local[((((ff_c_init * 4) + xx_c_init) + 48))] = 0.000000e+00f;
    }
  }
  for (int rc_outer = 0; rc_outer < 16; ++rc_outer) {
    for (int ry_outer = 0; ry_outer < 5; ++ry_outer) {
      __syncthreads();
      for (int mfi = 0; mfi < 5; ++mfi) {
        pad_temp_shared[ry_outer * 320 + (((((threadIdx.z * 40) + (threadIdx.y * 20)) + (threadIdx.x * 5)) + mfi))] = (((((2 <= (((blockIdx.y * 8) + ry_outer) + (((threadIdx.z * 2) + threadIdx.y) & 7))) && ((((blockIdx.y * 8) + ry_outer) + (((threadIdx.z * 2) + threadIdx.y) & 7)) < 258)) && (2 <= (((blockIdx.x * 16) + (threadIdx.x * 5)) + mfi))) && ((((blockIdx.x * 16) + (threadIdx.x * 5)) + mfi) < 258)) ? data[((((((((((rc_outer * 131072) + ((((threadIdx.z * 2) + threadIdx.y) >> 3) * 65536)) + (blockIdx.y * 2048)) + (ry_outer * 256)) + ((((threadIdx.z * 2) + threadIdx.y) & 7) * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 5)) + mfi) - 514))] : 0.000000e+00f);
      }
    }

    for (int ry_outer = 0; ry_outer < 5; ++ry_outer) {
      __syncthreads();
      for (int mfi1 = 0; mfi1 < 5; ++mfi1) {
        kernel5_shared[(((((threadIdx.z * 40) + (threadIdx.y * 20)) + (threadIdx.x * 5)) + mfi1))] = kernel5[((((((((threadIdx.z * 3200) + (threadIdx.y * 1600)) + ((threadIdx.x >> 1) * 800)) + (rc_outer * 50)) + ((threadIdx.x & 1) * 25)) + (ry_outer * 5)) + mfi1))];
      }
      __syncthreads();
      for (int rc_io = 0; rc_io < 2; ++rc_io) {
        for (int rx_io = 0; rx_io < 5; ++rx_io) {
          for (int ax3 = 0; ax3 < 4; ++ax3) {
            pad_temp_shared_local[(ax3)] = pad_temp_shared[ry_outer*320 + ((((((rc_io * 160) + (threadIdx.y * 20)) + (threadIdx.x * 4)) + ax3) + rx_io))];
            pad_temp_shared_local[((ax3 + 4))] = pad_temp_shared[ry_outer*320 + (((((((rc_io * 160) + (threadIdx.y * 20)) + (threadIdx.x * 4)) + ax3) + rx_io) + 40))];
            pad_temp_shared_local[((ax3 + 8))] = pad_temp_shared[ry_outer*320 + (((((((rc_io * 160) + (threadIdx.y * 20)) + (threadIdx.x * 4)) + ax3) + rx_io) + 80))];
            pad_temp_shared_local[((ax3 + 12))] = pad_temp_shared[ry_outer*320 + (((((((rc_io * 160) + (threadIdx.y * 20)) + (threadIdx.x * 4)) + ax3) + rx_io) + 120))];
          }
          for (int ax0 = 0; ax0 < 4; ++ax0) {
            kernel_shared_local[(ax0)] = kernel5_shared[(((((threadIdx.z * 40) + (ax0 * 10)) + (rc_io * 5)) + rx_io))];
          }
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            for (int xx_c = 0; xx_c < 4; ++xx_c) {
              compute5_local[(((ff_c * 4) + xx_c))] = (compute5_local[(((ff_c * 4) + xx_c))] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(ff_c)]));
              compute5_local[((((ff_c * 4) + xx_c) + 16))] = (compute5_local[((((ff_c * 4) + xx_c) + 16))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(ff_c)]));
              compute5_local[((((ff_c * 4) + xx_c) + 32))] = (compute5_local[((((ff_c * 4) + xx_c) + 32))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(ff_c)]));
              compute5_local[((((ff_c * 4) + xx_c) + 48))] = (compute5_local[((((ff_c * 4) + xx_c) + 48))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(ff_c)]));
            }
          }
        }
      }
    }

    for (int ry_outer = 0; ry_outer < 3; ++ry_outer) {
      __syncthreads();
      for (int mfi1 = 0; mfi1 < 3; ++mfi1) {
        kernel3_shared[(((((threadIdx.z * 24) + (threadIdx.y * 12)) + (threadIdx.x * 3)) + mfi1))] = kernel3[((((((((threadIdx.z * 1152) + (threadIdx.y * 576)) + ((threadIdx.x >> 1) * 288)) + (rc_outer * 18)) + ((threadIdx.x & 1) * 9)) + (ry_outer * 3)) + mfi1))];
      }
      __syncthreads();
      for (int rc_io = 0; rc_io < 2; ++rc_io) {
        for (int rx_io = 0; rx_io < 3; ++rx_io) {
          for (int ax3 = 0; ax3 < 4; ++ax3) {
            pad_temp_shared_local[(ax3)] = pad_temp_shared[ry_outer*192 + ((((((rc_io * 96) + (threadIdx.y * 12)) + (threadIdx.x * 4)) + ax3) + rx_io))];
            pad_temp_shared_local[((ax3 + 4))] = pad_temp_shared[ry_outer*192 + (((((((rc_io * 96) + (threadIdx.y * 12)) + (threadIdx.x * 4)) + ax3) + rx_io) + 24))];
            pad_temp_shared_local[((ax3 + 8))] = pad_temp_shared[ry_outer*192 + (((((((rc_io * 96) + (threadIdx.y * 12)) + (threadIdx.x * 4)) + ax3) + rx_io) + 48))];
            pad_temp_shared_local[((ax3 + 12))] = pad_temp_shared[ry_outer*192 + (((((((rc_io * 96) + (threadIdx.y * 12)) + (threadIdx.x * 4)) + ax3) + rx_io) + 72))];
          }
          for (int ax0 = 0; ax0 < 4; ++ax0) {
            kernel_shared_local[(ax0)] = kernel3_shared[(((((threadIdx.z * 24) + (ax0 * 6)) + (rc_io * 3)) + rx_io))];
          }
          for (int ff_c = 0; ff_c < 4; ++ff_c) {
            for (int xx_c = 0; xx_c < 4; ++xx_c) {
              compute3_local[(((ff_c * 4) + xx_c))] = (compute3_local[(((ff_c * 4) + xx_c))] + (pad_temp_shared_local[(xx_c)] * kernel_shared_local[(ff_c)]));
              compute3_local[((((ff_c * 4) + xx_c) + 16))] = (compute3_local[((((ff_c * 4) + xx_c) + 16))] + (pad_temp_shared_local[((xx_c + 4))] * kernel_shared_local[(ff_c)]));
              compute3_local[((((ff_c * 4) + xx_c) + 32))] = (compute3_local[((((ff_c * 4) + xx_c) + 32))] + (pad_temp_shared_local[((xx_c + 8))] * kernel_shared_local[(ff_c)]));
              compute3_local[((((ff_c * 4) + xx_c) + 48))] = (compute3_local[((((ff_c * 4) + xx_c) + 48))] + (pad_temp_shared_local[((xx_c + 12))] * kernel_shared_local[(ff_c)]));
            }
          }
        }
      }
    }
  }
  for (int ff_iii = 0; ff_iii < 4; ++ff_iii) {
    for (int xx_iii = 0; xx_iii < 4; ++xx_iii) {
      compute5[((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii))] = compute5_local[(((ff_iii * 4) + xx_iii))];
      compute5[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 512))] = compute5_local[((((ff_iii * 4) + xx_iii) + 16))];
      compute5[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 1024))] = compute5_local[((((ff_iii * 4) + xx_iii) + 32))];
      compute5[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 1536))] = compute5_local[((((ff_iii * 4) + xx_iii) + 48))];


      compute3[((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii))] = compute3_local[(((ff_iii * 4) + xx_iii))];
      compute3[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 512))] = compute3_local[((((ff_iii * 4) + xx_iii) + 16))];
      compute3[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 1024))] = compute3_local[((((ff_iii * 4) + xx_iii) + 32))];
      compute3[(((((((((threadIdx.z * 262144) + (ff_iii * 65536)) + (blockIdx.y * 2048)) + (threadIdx.y * 256)) + (blockIdx.x * 16)) + (threadIdx.x * 4)) + xx_iii) + 1536))] = compute3_local[((((ff_iii * 4) + xx_iii) + 48))];
    }
  }
}

extern "C" __global__ void conv_fused(float* __restrict__ data,
				      float* __restrict__ kernel,
				      float* __restrict__ compute) {
  float compute_local[16];
  __shared__ float pad_temp_shared[640];
  __shared__ float kernel_shared[3200];
  float pad_temp_shared_local[16];
  float kernel_shared_local[1];
  compute_local[(0)] = 0.000000e+00f;
  compute_local[(1)] = 0.000000e+00f;
  compute_local[(2)] = 0.000000e+00f;
  compute_local[(3)] = 0.000000e+00f;
  compute_local[(4)] = 0.000000e+00f;
  compute_local[(5)] = 0.000000e+00f;
  compute_local[(6)] = 0.000000e+00f;
  compute_local[(7)] = 0.000000e+00f;
  compute_local[(8)] = 0.000000e+00f;
  compute_local[(9)] = 0.000000e+00f;
  compute_local[(10)] = 0.000000e+00f;
  compute_local[(11)] = 0.000000e+00f;
  compute_local[(12)] = 0.000000e+00f;
  compute_local[(13)] = 0.000000e+00f;
  compute_local[(14)] = 0.000000e+00f;
  compute_local[(15)] = 0.000000e+00f;
  for (int rc_outer = 0; rc_outer < 8; ++rc_outer) {
    __syncthreads();
    pad_temp_shared[(((((int)threadIdx.z) * 20) + (((int)threadIdx.x) * 5)))] = ((((2 <= ((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7))) && (((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7)) < 258)) && (2 <= ((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 5)))) ? data[((((((((rc_outer * 262144) + ((((int)threadIdx.z) >> 3) * 65536)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.z) & 7) * 256)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 5)) - 514))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 20) + (((int)threadIdx.x) * 5)) + 1))] = ((((2 <= ((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7))) && (((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7)) < 258)) && (1 <= ((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 5)))) ? data[((((((((rc_outer * 262144) + ((((int)threadIdx.z) >> 3) * 65536)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.z) & 7) * 256)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 5)) - 513))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 20) + (((int)threadIdx.x) * 5)) + 2))] = (((2 <= ((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7))) && (((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7)) < 258)) ? data[((((((((rc_outer * 262144) + ((((int)threadIdx.z) >> 3) * 65536)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.z) & 7) * 256)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 5)) - 512))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 20) + (((int)threadIdx.x) * 5)) + 3))] = ((((2 <= ((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7))) && (((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7)) < 258)) && (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 5)) < 255)) ? data[((((((((rc_outer * 262144) + ((((int)threadIdx.z) >> 3) * 65536)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.z) & 7) * 256)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 5)) - 511))] : 0.000000e+00f);
    pad_temp_shared[((((((int)threadIdx.z) * 20) + (((int)threadIdx.x) * 5)) + 4))] = ((((2 <= ((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7))) && (((((int)blockIdx.y) * 4) + (((int)threadIdx.z) & 7)) < 258)) && (((((int)blockIdx.x) * 16) + (((int)threadIdx.x) * 5)) < 254)) ? data[((((((((rc_outer * 262144) + ((((int)threadIdx.z) >> 3) * 65536)) + (((int)blockIdx.y) * 1024)) + ((((int)threadIdx.z) & 7) * 256)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 5)) - 510))] : 0.000000e+00f);
    kernel_shared[(((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)))] = kernel[(((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 1))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 1))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 2))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 2))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 3))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 3))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 4))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 4))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 5))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 5))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 6))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 6))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 7))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 7))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 8))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 8))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 9))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 9))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 10))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 10))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 11))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 11))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 12))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 12))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 13))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 13))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 14))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 14))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 15))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 15))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 16))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 16))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 17))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 17))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 18))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 18))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 19))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 19))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 20))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 20))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 21))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 21))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 22))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 22))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 23))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 23))];
    kernel_shared[((((((int)threadIdx.z) * 100) + (((int)threadIdx.x) * 25)) + 24))] = kernel[((((((((int)blockIdx.z) * 25600) + (((int)threadIdx.z) * 800)) + (rc_outer * 100)) + (((int)threadIdx.x) * 25)) + 24))];
    __syncthreads();
    for (int rc_inner_outer = 0; rc_inner_outer < 4; ++rc_inner_outer) {
      pad_temp_shared_local[(0)] = pad_temp_shared[(((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 1))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 2))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 3))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 20))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 21))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 40))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 60))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      kernel_shared_local[(0)] = kernel_shared[(((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 1))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 2))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 3))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 4))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 21))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 1))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 2))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 3))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 4))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 5))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 2))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 3))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 4))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 5))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 6))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 26))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 3))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 4))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 5))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 6))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 7))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 26))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 27))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 47))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 67))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 4))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 20))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 21))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 40))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 60))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 80))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 5))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 21))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 6))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 22))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 7))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 23))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 26))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 8))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 24))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 25))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 26))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 27))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 47))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 67))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 87))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 9))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 40))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 60))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 80))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 100))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 10))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 41))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 11))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 42))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 12))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 43))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 13))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 44))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 45))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 46))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 47))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 67))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 87))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 107))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 14))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 60))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 80))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 100))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 120))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 121))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 15))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 61))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 121))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 16))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 62))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 17))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 63))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 126))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 18))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 64))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 65))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 66))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 67))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 87))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 107))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 126))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 127))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 19))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 80))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 100))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 120))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 121))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 140))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 141))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 142))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 143))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 20))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 81))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 101))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 121))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 141))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 142))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 143))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 144))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 21))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 82))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 102))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 122))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 142))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 143))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 144))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 145))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 22))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 83))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 103))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 123))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 126))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 143))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 144))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 145))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 146))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 23))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
      pad_temp_shared_local[(0)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 84))];
      pad_temp_shared_local[(1)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 85))];
      pad_temp_shared_local[(2)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 86))];
      pad_temp_shared_local[(3)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 87))];
      pad_temp_shared_local[(4)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 104))];
      pad_temp_shared_local[(5)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 105))];
      pad_temp_shared_local[(6)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 106))];
      pad_temp_shared_local[(7)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 107))];
      pad_temp_shared_local[(8)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 124))];
      pad_temp_shared_local[(9)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 125))];
      pad_temp_shared_local[(10)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 126))];
      pad_temp_shared_local[(11)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 127))];
      pad_temp_shared_local[(12)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 144))];
      pad_temp_shared_local[(13)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 145))];
      pad_temp_shared_local[(14)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 146))];
      pad_temp_shared_local[(15)] = pad_temp_shared[((((rc_inner_outer * 160) + (((int)threadIdx.x) * 4)) + 147))];
      kernel_shared_local[(0)] = kernel_shared[((((((int)threadIdx.z) * 100) + (rc_inner_outer * 25)) + 24))];
      compute_local[(0)] = (compute_local[(0)] + (pad_temp_shared_local[(0)] * kernel_shared_local[(0)]));
      compute_local[(1)] = (compute_local[(1)] + (pad_temp_shared_local[(1)] * kernel_shared_local[(0)]));
      compute_local[(2)] = (compute_local[(2)] + (pad_temp_shared_local[(2)] * kernel_shared_local[(0)]));
      compute_local[(3)] = (compute_local[(3)] + (pad_temp_shared_local[(3)] * kernel_shared_local[(0)]));
      compute_local[(4)] = (compute_local[(4)] + (pad_temp_shared_local[(4)] * kernel_shared_local[(0)]));
      compute_local[(5)] = (compute_local[(5)] + (pad_temp_shared_local[(5)] * kernel_shared_local[(0)]));
      compute_local[(6)] = (compute_local[(6)] + (pad_temp_shared_local[(6)] * kernel_shared_local[(0)]));
      compute_local[(7)] = (compute_local[(7)] + (pad_temp_shared_local[(7)] * kernel_shared_local[(0)]));
      compute_local[(8)] = (compute_local[(8)] + (pad_temp_shared_local[(8)] * kernel_shared_local[(0)]));
      compute_local[(9)] = (compute_local[(9)] + (pad_temp_shared_local[(9)] * kernel_shared_local[(0)]));
      compute_local[(10)] = (compute_local[(10)] + (pad_temp_shared_local[(10)] * kernel_shared_local[(0)]));
      compute_local[(11)] = (compute_local[(11)] + (pad_temp_shared_local[(11)] * kernel_shared_local[(0)]));
      compute_local[(12)] = (compute_local[(12)] + (pad_temp_shared_local[(12)] * kernel_shared_local[(0)]));
      compute_local[(13)] = (compute_local[(13)] + (pad_temp_shared_local[(13)] * kernel_shared_local[(0)]));
      compute_local[(14)] = (compute_local[(14)] + (pad_temp_shared_local[(14)] * kernel_shared_local[(0)]));
      compute_local[(15)] = (compute_local[(15)] + (pad_temp_shared_local[(15)] * kernel_shared_local[(0)]));
    }
  }
  compute[((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)))] = compute_local[(0)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 1))] = compute_local[(1)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 2))] = compute_local[(2)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 3))] = compute_local[(3)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 256))] = compute_local[(4)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 257))] = compute_local[(5)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 258))] = compute_local[(6)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 259))] = compute_local[(7)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 512))] = compute_local[(8)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 513))] = compute_local[(9)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 514))] = compute_local[(10)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 515))] = compute_local[(11)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 768))] = compute_local[(12)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 769))] = compute_local[(13)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 770))] = compute_local[(14)];
  compute[(((((((((int)blockIdx.z) * 2097152) + (((int)threadIdx.z) * 65536)) + (((int)blockIdx.y) * 1024)) + (((int)blockIdx.x) * 16)) + (((int)threadIdx.x) * 4)) + 771))] = compute_local[(15)];
}

int main() {
  int batch = 1;
  int in_channel = 32;
  int in_height = 256;
  int in_width = 256;
  int num_filter = 64;

  float* images;
  float* kernelsf;
  float* computef;
  float* kernels5;
  float* compute5;
  float* kernels3;
  float* compute3;

  hipMalloc((void**)&images, batch * in_channel * in_height * in_width * sizeof(float));

  hipMalloc((void**)&kernelsf, 2 * num_filter * in_channel * 5 * 5 * sizeof(float));
  hipMalloc((void**)&computef, 2 * batch * in_width * in_height * num_filter * sizeof(float));

  hipMalloc((void**)&kernels5, num_filter * in_channel * 5 * 5 * sizeof(float));
  hipMalloc((void**)&compute5, batch * in_width * in_height * num_filter * sizeof(float));

  hipMalloc((void**)&kernels3, 2 * num_filter * in_channel * 3 * 3 * sizeof(float));
  hipMalloc((void**)&compute3, batch * in_width * in_height * num_filter * sizeof(float));

  int w_iters = 1000;
  int iters = 1000 + w_iters;
  float no_fused5 = 0;
  float no_fused3 = 0;
  float partially_fused = 0;
  float fully_fused = 0;

  dim3 ugrid3 = dim3(16, 32, 1);
  dim3 ublock3 = dim3(16, 1, 8);
  dim3 ugrid5 = dim3(4, 64, 1);
  dim3 ublock5 = dim3(16, 1, 8);

  // dim3 pgrid = dim3(16, 32, 1);
  // dim3 pblock = dim3(4, 2, 8);
  dim3 pgrid = dim3(4, 64, 1);
  dim3 pblock = dim3(16, 1, 8);

  dim3 fgrid = dim3(16, 64, 2);
  dim3 fblock = dim3(4, 1, 32);

  for (int i = 0; i < iters; ++i) {
    {
      hipEvent_t start, end;
      float elapsed = 0;
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      conv5<<<ugrid5, ublock5>>>(images, kernels5, compute5);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      if (i >= w_iters) no_fused5 += elapsed;
    }
    {
      hipEvent_t start, end;
      float elapsed = 0;
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      conv3<<<ugrid3, ublock3>>>(images, kernels3, compute3);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      if (i >= w_iters) no_fused3 += elapsed;
    }
    {
      hipEvent_t start, end;
      float elapsed = 0;
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      conv_partially_fused<<<pgrid, pblock>>>(images, kernels5, kernels3, compute5, compute3);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      if (i >= w_iters) partially_fused += elapsed;
    }
    {
      hipEvent_t start, end;
      float elapsed = 0;
      hipEventCreate(&start);
      hipEventCreate(&end);
      hipEventRecord(start);

      conv_fused<<<fgrid, fblock>>>(images, kernelsf, computef);

      hipEventRecord(end);
      hipEventSynchronize(end);
      hipEventElapsedTime(&elapsed, start, end);
      if (i >= w_iters) fully_fused += elapsed;
    }
  }

  float no_fused = no_fused5 + no_fused3;
  std::cout << "No Fusion      : " << no_fused / iters << " (" << no_fused5 / iters << ", " << no_fused3 / iters << ")" << std::endl;
  std::cout << "Partial Fusion : " << partially_fused / iters << std::endl;
  std::cout << "Full Fusion    : " << fully_fused / iters << std::endl;
}
